#include "hip/hip_runtime.h"
#include "MatrixTransposeSolveBankConflicts.h"
#include "config.h"

__global__ void matrixTransposeSolveBankConflicts(const int *d_a, int *d_b, const int rows, const int cols) {

    __shared__ int mat[BLOCK_SIZE][BLOCK_SIZE + 1];

    int bx = blockIdx.x * BLOCK_SIZE;
    int by = blockIdx.y * BLOCK_SIZE;

    int i = by + threadIdx.y; int j = bx + threadIdx.x;
    int ti = bx + threadIdx.y; int tj = by + threadIdx.x;

    if (i<rows && j<cols)
        mat[threadIdx.y][threadIdx.x] = d_a[i*cols+j];

    __syncthreads();
    if (tj < cols && ti<rows)
        d_b[ti*rows+tj]=mat[threadIdx.x][threadIdx.y];
}
